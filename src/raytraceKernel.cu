#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  ray r;

  	glm::vec3 A,B,M,H,V,P,D ;

	// Calculating the unknown angel theta
float angelv = tan(float(fov.y/57.295779));
float EtoM = (resolution.y/2)/angelv;
float Fovx = atan(float ((resolution.x/2)/EtoM) )*57.295779;
float angelh = tan(float((Fovx)/57.295779));

// Finding the unknown parameters that are required for ray march
A = glm::cross(view,up); 
B = glm::cross(A,view);
M = eye + view;
H = (glm::normalize (A))* sqrt(view.x*view.x + view.y*view.y +view.z*view.z )*(angelh) ;
V = (glm::normalize( B))* sqrt(view.x*view.x + view.y*view.y +view.z*view.z )*(angelv) ; 
float sx,sy ;

 sx=(x/float (resolution.x-1));
 sy=(float(resolution.y-1-y)/float(resolution.y-1)); // To make sure easy BMP displays properly the formula is modified.
 P = M + (((2*sx)-1)*H) + (((2*sy)-1)*V); 
 glm::vec3 aa = P - eye;
 D = glm::normalize(aa) ;



  r.origin = eye;
  r.direction = D;
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms,material* cudamat ,int numberOfMaterials,glm::vec3* myvertex, int numVertices){          //

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  int p = numberOfGeoms;
  glm::vec3 norms[10],ips[10];
  float rips[10];

  
  //Hard coded triangle vertices
  glm::vec3 p11(0,0.5,0);
  glm::vec3 p12(1,-0.5,0);
  glm::vec3 p13(-1,-0.5,0);

  int obno=0;
  float inf = 123456.0;
  glm::vec3 dnorm, dips, tnorm, tips;
  float dhit = 123456.0,thit ;

  if((x<=resolution.x && y<=resolution.y)){
  
  ray r = raycastFromCameraKernel(resolution,time,x,y,cam.position,cam.view,cam.up,cam.fov);
  
  for(int i=0; i < numberOfGeoms ; i++)
  {
		// Intersection tests with different objects
		if(geoms[i].type == SPHERE)
		{
		  thit = sphereIntersectionTest(geoms[i],r, tips, tnorm);
		}  
		else if (geoms[i].type == CUBE)
		{
		  thit = boxIntersectionTest(geoms[i], r, tips, tnorm);
		}
		else if (geoms[i].type == MESH)
		{ 
		  thit = meshIntersectionTest(geoms[i],r,myvertex,numVertices, tips, tnorm);
		}

		//Finding the closest of the intersection points
		if(thit == -1)
			thit = 123456.0;

		if(thit <= dhit)
		{
			dhit  = thit;
			dips  = tips;
			dnorm = tnorm;
			obno  = i; 
		}

  }

if( dhit == 123456.0)
	{
		colors[index] = glm::vec3(0,0,0);
	}
else
{
	  glm::vec3 LPOS = glm::vec3(0,8.5,4);

	  // Calculating the specular component
	    glm::vec3 lig =  dips - LPOS ;
		glm::vec3 ref1  =  lig - (2.0f * dnorm * (glm::dot(dnorm,lig)));     // glm::normalize(glm::reflect(-(lig) , glm::normalize(norms[obno])));
		float dt = glm::dot(glm::normalize(cam.position),glm::normalize(ref1));
		if(dt < 0)
			 dt = 0;
		 float sc = pow(dt,30);


	   // Shadows
	   int shadow = 1 ;
	 //  float sit = 0; ;
	  glm::vec3 neyep = dips + ref1 * 0.001f ;
	  ray s;
	  s.origin = neyep;
	  s.direction = glm::normalize(LPOS-neyep);
	  shadow = checkForShadows(geoms,numberOfGeoms,s,myvertex,numVertices,LPOS,obno);
	

	// Reflections ////////////////////////////////////////////////////////////////////

		ray ref;
		ref.origin = neyep;
		ref.direction = ref1 ;
		glm::vec3 rips,rnorm;
		int rbno = -1;
        if(cudamat[geoms[obno].materialid].hasReflective != 0  &&  shadow != 1)
		 rbno = getreflectedcolor(geoms,numberOfGeoms,ref,myvertex,numVertices,rips,rnorm);
	
		glm::vec3 relcolor(0,0,0);
		if(rbno != -1 )
		{
			relcolor = cudamat[geoms[rbno].materialid].color *cudamat[geoms[rbno].materialid].hasReflective ; //
		}
	   


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //(LCOL * pnod->color * (dot(N,normalize(LPOS - ipoint))))
	   //float ab = glm::dot(norms[obno],glm::normalize(LPOS - ips[obno]));
		float kd = 0.75f,ks = 0.2f,ka = 0.05;
		glm::vec3 amb = ka * cudamat[geoms[obno].materialid].color;
		if(shadow == 0)
		{
  //  colors[index] = color[geoms[obno].materialid] *  glm::dot(norms[obno],glm::normalize(LPOS - ips[obno]))  ;// +   glm::vec3(1,1,1) * sc   + relcolor;   //glm::vec3(1,1,1) * 
		//colors[index] = cudamat[geoms[obno].materialid].color *  glm::dot(norms[obno],glm::normalize(LPOS - ips[obno])) ;//+   glm::vec3(1,1,1) * sc  ;
		colors[index] =(amb + kd * cudamat[geoms[obno].materialid].color *  glm::dot(dnorm,glm::normalize(LPOS - dips)) + ks *glm::vec3(1,1,1) * sc)*(1-ks)  + ks * relcolor  ;
		//colors[index] = cudamat[geoms[obno].materialid].color *  glm::dot(dnorm,glm::normalize(LPOS - dips)) + glm::vec3(1,1,1) * sc;
		}
		else
		{
		colors[index]  =  glm::vec3(0,0,0);//cudamat[geoms[obno].materialid].color * 0.1f; //
		}
}
  //colors[index] = glm::vec3(fabsf(r.direction.x),fabsf(r.direction.y),fabsf(r.direction.z));
 
  }

  

 //colors[index] = generateRandomNumberFromThread(resolution, time, x, y);
   
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms,std::vector<glm::vec3> mypoints){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  int numVertices = mypoints.size();
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //Send vertices of the mesh to GPU
  glm::vec3* mvertex = NULL;
  hipMalloc((void**)&mvertex,mypoints.size() * sizeof(glm::vec3));
  for(int i=0; i < mypoints.size(); i++){
	   
	   hipMemcpy( &mvertex[i] , &mypoints[i], sizeof(glm::vec3), hipMemcpyHostToDevice);
  }

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  // package materials and send it to GPU
   material* matList = new material[numberOfMaterials];
  for(int i=0; i<numberOfMaterials; i++){
    material newMaterial;
    newMaterial.color = materials[i].color;
    newMaterial.specularExponent = materials[i].specularExponent;
    newMaterial.specularColor = materials[i].specularColor;
	newMaterial.hasReflective = materials[i].hasReflective;
    newMaterial.hasRefractive = materials[i].hasRefractive;
	newMaterial.indexOfRefraction = materials[i].indexOfRefraction;
	newMaterial.hasScatter = materials[i].hasScatter;
	newMaterial.absorptionCoefficient = materials[i].absorptionCoefficient;
	newMaterial.reducedScatterCoefficient = materials[i].reducedScatterCoefficient;
	newMaterial.emittance = materials[i].emittance;
	matList[i] = newMaterial;
  }
  
  material* cudamat = NULL;
  hipMalloc((void**)&cudamat, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamat, matList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //package material color
  glm::vec3* color = NULL;
  hipMalloc((void**)&color, numberOfMaterials*sizeof(glm::vec3));
   for(int i=0;i <  numberOfMaterials ; i++)
   {
   glm::vec3* tcolor = color + i ;
  hipMemcpy(tcolor , &materials[i].color, sizeof(glm::vec3), hipMemcpyHostToDevice);
  }
  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms,cudamat ,numberOfMaterials,mvertex,numVertices);  //

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

float __device__ meshIntersectionTest(staticGeom curGeom,ray s,glm::vec3* myvertex, int numVertices, glm::vec3& mintersect, glm::vec3& mnormal)
{
				int flag = 0 ;
				glm::vec3 ipss,normss;
			float t , at[100];
			int p = 0;
			glm::vec3 curnorm[100] , curipss[100];
		for(int k=0 ;k < numVertices - 2 ; k= k+3)          
		{
			t = triangleIntersectionTest(curGeom,s,myvertex[k],myvertex[k+1],myvertex[k+2], ipss, normss);
			if(t != -1)
			{
				curnorm[p]  = normss;
				curipss[p]  = ipss;
				flag = 1;
				at[p] = t;
				p++;
			}
		}  //colors[index] = materials[geoms[i].materialid].color;
		float ntemp = at[0];   
		int nindex = 0;
		if(flag == 1)
		{
			for(int s =0 ; s < p ; s++)
			{
				if(at[s] < ntemp)
				{
					ntemp = at[s];
					nindex = s;
				}
			 // triangleIntersectionTest(geoms[i],r,p11,p12,p13, ips[i], norms[i]);
			
			}

			
			mnormal    = curnorm[nindex];
			mintersect = curipss[nindex];
			return  at[nindex] ;
		}
		else
			return -1;

}


int __device__ checkForShadows(staticGeom* geoms,int numberOfGeoms,ray s, glm::vec3* myvertex, int numVertices,glm::vec3 LPOS,int obno)
{
	   int sha = 0;
	   float sit = 0;
	   float len = glm::length(s.origin - LPOS) ;
	  
	   glm::vec3 htemp,ntemp;
	    for(int i=0 ; i <numberOfGeoms ; i++)
        {
		if(i != -1)
		{
			if(geoms[i].type == SPHERE)
		{
		  sit = sphereIntersectionTest(geoms[i],s, htemp, ntemp);
		}  
		else if (geoms[i].type == CUBE)
		{
		  sit = boxIntersectionTest(geoms[i],s,  htemp, ntemp);
		}
		else if (geoms[i].type == MESH)
		{
		  sit =  meshIntersectionTest(geoms[i],s,myvertex,numVertices,htemp,ntemp);
		}
       
	     if(sit != -1)
	       {
			if ( glm::length(htemp - s.origin ) < len)
				return 1;   //  Shadow == 1 means the point of interesection is under a shadow , if 0 then no shadow
			else
				sha = 0 ;
		 }
		}	 
		}

		return sha;
}


int __device__ getreflectedcolor(staticGeom* geoms,int numberOfGeoms,ray ref, glm::vec3* myvertex, int numVertices,glm::vec3& htemp, glm::vec3& ntemp)
{
	glm::vec3 rips,rnorm,trips,trnorm;
	float rhit = 123456.0,trhit ;
	int rno = -1;
	for(int i=0; i < numberOfGeoms ; i++)
  {
		// Intersection tests with different objects
		if(geoms[i].type == SPHERE)
		{
		  trhit = sphereIntersectionTest(geoms[i],ref, trips,trnorm);
		}  
		else if (geoms[i].type == CUBE)
		{
		  trhit = boxIntersectionTest(geoms[i], ref, trips,trnorm);
		}
		else if (geoms[i].type == MESH)
		{ 
		  trhit = meshIntersectionTest(geoms[i],ref,myvertex,numVertices, trips,trnorm);
		}

		//Finding the closest of the intersection points
		if(trhit == -1)
			trhit = 123456.0;

		if(trhit <= rhit)
		{
			rhit  = trhit;
			htemp = trips;
			ntemp = trnorm;
			rno   = i; 
		}

  }

	return rno;


}
