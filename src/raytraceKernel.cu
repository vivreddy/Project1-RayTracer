#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  ray r;

  	glm::vec3 A,B,M,H,V,P,D ;

	// Calculating the unknown angel theta
float angelv = tan(float(fov.y/57.295779));
float EtoM = (resolution.y/2)/angelv;
float Fovx = atan(float ((resolution.x/2)/EtoM) )*57.295779;
float angelh = tan(float((Fovx)/57.295779));

// Finding the unknown parameters that are required for ray march
A = glm::cross(view,up); 
B = glm::cross(A,view);
M = eye + view;
H = (glm::normalize (A))* sqrt(view.x*view.x + view.y*view.y +view.z*view.z )*(angelh) ;
V = (glm::normalize( B))* sqrt(view.x*view.x + view.y*view.y +view.z*view.z )*(angelv) ; 
float sx,sy ;

 sx=(x/float (resolution.x-1));
 sy=(float(resolution.y-1-y)/float(resolution.y-1)); // To make sure easy BMP displays properly the formula is modified.
 P = M + (((2*sx)-1)*H) + (((2*sy)-1)*V); 
 glm::vec3 aa = P - eye;
 D = glm::normalize(aa) ;



  r.origin = eye;
  r.direction = D;
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms,material* cudamat ,int numberOfMaterials,glm::vec3* myvertex, int numVertices){          //

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  //Initializing variables for intersection
  int obno=0;
  float inf = 123456.0;
  glm::vec3 dnorm, dips, tnorm, tips;
  float dhit = 123456.0,thit ;
  int glid;

  if((x<=resolution.x && y<=resolution.y)){
  
  ray r = raycastFromCameraKernel(resolution,time,x,y,cam.position,cam.view,cam.up,cam.fov);
  
  // Finding the interection with geometrys
  for(int i=0; i < numberOfGeoms ; i++)
  {
		// Intersection tests with different objects
		if(geoms[i].type == SPHERE)
		{
		  thit = sphereIntersectionTest(geoms[i],r, tips, tnorm);
		}  
		else if (geoms[i].type == CUBE)
		{
		  thit = boxIntersectionTest(geoms[i], r, tips, tnorm);
		}
		else if (geoms[i].type == MESH)
		{ 
		  thit = meshIntersectionTest(geoms[i],r,myvertex,numVertices, tips, tnorm);
		}

		//Finding the closest of the intersection points
		if(thit == -1)
			thit = 123456.0;

		if(thit <= dhit)
		{
			dhit  = thit;
			dips  = tips;
			dnorm = tnorm;
			obno  = i; 
		}

		if(geoms[i].materialid == 8)
			glid = i;
  }

	if( dhit == 123456.0)
	{
		//Output the background color
		colors[index] = glm::vec3(0,0,0);
	}
	else
	{
	   glm::vec3 LPOS = glm::vec3(0,9.5,0.0); // multiplyMV(geoms[glid].transform, glm::vec4(glm::vec3(0,-1,0),1.0f));// + glm::vec3(0,-2.0,0); //glm::vec3(0,8.5,0);

	 //   // Calculating the specular component
	    glm::vec3 lig =  dips - LPOS ;
		glm::vec3 ref1  =  lig - (2.0f * dnorm * (glm::dot(dnorm,lig)));     // glm::normalize(glm::reflect(-(lig) , glm::normalize(norms[obno])));
		//float dt = glm::dot(glm::normalize(cam.position),glm::normalize(ref1));
		//if(dt < 0)
		//	 dt = 0;
		//float sc = pow(dt,30);

		// Shadows
	    int shadow = 1 ;
		glm::vec3 neyep = dips + ref1 * 0.001f ;
		ray s;
		s.origin = neyep;
		/*s.direction = glm::normalize(LPOS-neyep);
		shadow = checkForShadows(geoms,numberOfGeoms,s,myvertex,numVertices,LPOS,obno);*/

		// Reflections 
		ray ref;
		ref.origin = neyep;
		ref.direction = ref1 ;
		glm::vec3 relcolor(0,0,0);
		glm::vec3 rips,rnorm;
		int rbno = -1;
        if(cudamat[geoms[obno].materialid].hasReflective != 0  ) //&&  shadow != 1
		{
			rbno = getreflectedcolor(geoms,numberOfGeoms,ref,myvertex,numVertices,rips,rnorm);
			if(rbno != -1 )
			 relcolor = cudamat[geoms[rbno].materialid].color;// *cudamat[geoms[rbno].materialid].hasReflective ;
		}
		
		if(rbno != -1 )
		{
			//relcolor = cudamat[geoms[rbno].materialid].color *cudamat[geoms[rbno].materialid].hasReflective ; //
		}
	   
		//Final output color
		float kd = 0.75f,ks = 0.2f,ka = 0.08;
		glm::vec3 amb = cudamat[geoms[obno].materialid].color;
		if(shadow == 0){
	//	colors[index] =(amb + kd * cudamat[geoms[obno].materialid].color *  glm::dot(dnorm,glm::normalize(LPOS - dips)) + ks *glm::vec3(1,1,1) * sc)*(1-ks)  + ks * relcolor  ;
		//colors[index] = cudamat[geoms[obno].materialid].color *  glm::dot(dnorm,glm::normalize(LPOS - dips)) + glm::vec3(1,1,1) * sc + + relcolor;
		}
		else{
	//	colors[index]  =  glm::vec3(0,0,0);//cudamat[geoms[obno].materialid].color * 0.1f; //
		}

	 glm::vec4 r1(2,0,0,0);
	 glm::vec4 r2(0,1,0,9.5);
	 glm::vec4 r3(0,0,2,0);
	 glm::vec4 r4(0,0,0,1);

	 cudaMat4 ittrans;
	 ittrans.x = r1;	
	 ittrans.y = r2;
     ittrans.z = r3;
	 ittrans.w = r4;

		//Soft shadows
		int dim = 20;
		glm::vec3 finalCol(0,0,0);
		float st = (1.0f/dim);
		float w = (1.0f/(dim*dim));
		glm::vec3 tlpos,mycolor;
		for(int i=0 ; i < dim ; i++)
		{
			for(int j=0 ; j < dim ; j++)
			{
			tlpos = glm::vec3(-0.5 + (st * i), 0, -0.5 + (st * j));
			LPOS = multiplyMV(ittrans, glm::vec4(tlpos,1.0f));//+ glm::vec3(0,-1.0,0);  //     geoms[glid].transform 
			//LPOS.y = 9.5;
			s.direction = glm::normalize(LPOS-neyep);
			shadow = checkForShadows(geoms,numberOfGeoms,s,myvertex,numVertices,LPOS,obno);
			calculateColoratPoint(geoms,dips,LPOS,dnorm,relcolor,obno,mycolor,cam.position,cudamat);
			if(shadow == 0)
				finalCol = finalCol + mycolor  ;
			else
				finalCol = finalCol + (amb * 0.1f); //(cudamat[geoms[obno].materialid].color *  glm::dot(dnorm,glm::normalize(LPOS - dips)))  ;//(amb * 0.4f);// +(amb * 0.1f) +* 0.1f
			}
		}
		if(cudamat[geoms[obno].materialid].emittance == 0)
		colors[index] = (finalCol * w) ; //(relcolor ); //
		else
		colors[index] = (finalCol * w) * 4.0f   ;
	}
  }

 //colors[index] = generateRandomNumberFromThread(resolution, time, x, y);   
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms,std::vector<glm::vec3> mypoints){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  int numVertices = mypoints.size();
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //Send vertices of the mesh to GPU
  glm::vec3* mvertex = NULL;
  hipMalloc((void**)&mvertex,mypoints.size() * sizeof(glm::vec3));
  for(int i=0; i < mypoints.size(); i++){
	   
	   hipMemcpy( &mvertex[i] , &mypoints[i], sizeof(glm::vec3), hipMemcpyHostToDevice);
  }

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  // package materials and send it to GPU
   material* matList = new material[numberOfMaterials];
  for(int i=0; i<numberOfMaterials; i++){
    material newMaterial;
    newMaterial.color = materials[i].color;
    newMaterial.specularExponent = materials[i].specularExponent;
    newMaterial.specularColor = materials[i].specularColor;
	newMaterial.hasReflective = materials[i].hasReflective;
    newMaterial.hasRefractive = materials[i].hasRefractive;
	newMaterial.indexOfRefraction = materials[i].indexOfRefraction;
	newMaterial.hasScatter = materials[i].hasScatter;
	newMaterial.absorptionCoefficient = materials[i].absorptionCoefficient;
	newMaterial.reducedScatterCoefficient = materials[i].reducedScatterCoefficient;
	newMaterial.emittance = materials[i].emittance;
	matList[i] = newMaterial;
  }
  
  material* cudamat = NULL;
  hipMalloc((void**)&cudamat, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamat, matList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //package material color
  glm::vec3* color = NULL;
  hipMalloc((void**)&color, numberOfMaterials*sizeof(glm::vec3));
   for(int i=0;i <  numberOfMaterials ; i++)
   {
   glm::vec3* tcolor = color + i ;
   hipMemcpy(tcolor , &materials[i].color, sizeof(glm::vec3), hipMemcpyHostToDevice);
   }

//create events
hipEvent_t event1, event2;
hipEventCreate(&event1);
hipEventCreate(&event2);

hipEventRecord(event1, 0); 

// Print time difference: ( end - begin )
  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms,cudamat ,numberOfMaterials,mvertex,numVertices);  //

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);


  hipEventRecord(event2, 0);
  //synchronize
hipEventSynchronize(event1); //optional
hipEventSynchronize(event2); //wait for the event to be executed!

//calculate time
float dt_ms;
hipEventElapsedTime(&dt_ms, event1, event2);

std::cout << dt_ms << std::endl ;
  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

float __device__ meshIntersectionTest(staticGeom curGeom,ray s,glm::vec3* myvertex, int numVertices, glm::vec3& mintersect, glm::vec3& mnormal)
{
		glm::vec3 ipss,normss;
		float t , at = 12345.0;
		glm::vec3 curnorm , curipss;

		for(int k=0 ;k < numVertices - 2 ; k= k+3)          
		{
			t = triangleIntersectionTest(curGeom,s,myvertex[k],myvertex[k+1],myvertex[k+2], ipss, normss);
			if(t != -1  && t<at)
			{
				curnorm  = normss;
				curipss  = ipss;
				at = t;
			}
		}  

		mnormal    = curnorm;
		mintersect = curipss;
		if (at == 12345.0)
			return -1;
		else
			return  at ;

}


int __device__ checkForShadows(staticGeom* geoms,int numberOfGeoms,ray s, glm::vec3* myvertex, int numVertices,glm::vec3 LPOS,int obno)
{
	   int   sha = 0;
	   float sit = 0;
	   float len = glm::length(s.origin - LPOS) ;
	  
	   glm::vec3 htemp,ntemp;
	    for(int i=0 ; i <numberOfGeoms ; i++)
        {
		if(i != -1)
		{
			if(geoms[i].type == SPHERE)
		{
		  sit = sphereIntersectionTest(geoms[i],s, htemp, ntemp);
		}  
		else if (geoms[i].type == CUBE)
		{
		  sit = boxIntersectionTest(geoms[i],s,  htemp, ntemp);
		}
		else if (geoms[i].type == MESH)
		{
		  sit =  meshIntersectionTest(geoms[i],s,myvertex,numVertices,htemp,ntemp);
		}
        //Shadow == 1 means the point of interesection is under a shadow , if 0 then no shadow
	    if(sit != -1)
	       {
			if ( glm::length(htemp - s.origin ) < len)
				return 1;   
			else
				sha = 0 ;
		   }
		}	 
		}

		return sha;
}


int __device__ getreflectedcolor(staticGeom* geoms,int numberOfGeoms,ray ref, glm::vec3* myvertex, int numVertices,glm::vec3& htemp, glm::vec3& ntemp)
{
	glm::vec3 rips,rnorm,trips,trnorm;
	float rhit = 123456.0,trhit ;
	int rno = -1;
	for(int i=0; i < numberOfGeoms ; i++)
  {
		// Intersection tests with different objects
		if(geoms[i].type == SPHERE)
		{
		  trhit = sphereIntersectionTest(geoms[i],ref, trips,trnorm);
		}  
		else if (geoms[i].type == CUBE)
		{
		  trhit = boxIntersectionTest(geoms[i], ref, trips,trnorm);
		}
		else if (geoms[i].type == MESH)
		{ 
		  trhit = meshIntersectionTest(geoms[i],ref,myvertex,numVertices, trips,trnorm);
		}

		//Finding the closest of the intersection points
		if(trhit == -1)
			trhit = 123456.0;

		if(trhit < rhit)
		{
			rhit  = trhit;
			htemp = trips;
			ntemp = trnorm;
			rno   = i; 
		}

  }

  return rno;


}


void __device__  calculateColoratPoint(staticGeom* geoms,glm::vec3 dips,glm::vec3 LPOS,glm::vec3 dnorm,glm::vec3 relcolor,int obno, glm::vec3& mycolor,glm::vec3 cpos,material* cudamat)
{

	    glm::vec3 lig =  glm::normalize(dips - LPOS) ;
		float sc = 0;
		glm::vec3 ref1  =  lig - (2.0f * dnorm * (glm::dot(dnorm,lig)));     // glm::normalize(glm::reflect(-(lig) , glm::normalize(norms[obno])));
		float dt = glm::dot(glm::normalize(cpos),glm::normalize(ref1));
		if(dt < 0)
			 dt = 0;
		if(cudamat[geoms[obno].materialid].specularExponent != 0)
			sc = pow(dt,cudamat[geoms[obno].materialid].specularExponent);

		//Final output color
		float kd = 0.9f,ks = 0.4f,kss = 0.2f,ka = 0.1;
		glm::vec3 amb =  cudamat[geoms[obno].materialid].color ;
		glm::vec3 dif =  cudamat[geoms[obno].materialid].color *  glm::dot(dnorm,glm::normalize(LPOS - dips));
		glm::vec3 spe =  glm::vec3(1,1,1) * sc ;
		glm::vec3 reff =  relcolor  ;

		//mycolor = (0.2f * amb + 0.6f* dif + 0.9f* spe) * 0.9f + (reff * 0.5f) ; //(ka * amb + kd * dif + kss * spe) * (1 - ks) + (ref * ks);
		mycolor = (kd * dif + 0.7f * spe) * 0.8f + reff * 0.2f ;
}